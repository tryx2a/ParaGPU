#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


int n = 200;
using namespace std;

__device__ float generate( hiprandState* globalState, int ind ) 
{
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    //float RANDOM = hiprand_uniform( &localState );
    float RANDOM = hiprand_normal( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void kernel(float* tabGauss, hiprandState* globalState, int n, int nbTours)
{
    // generate random numbers
    for(int i=0;i<nbTours;i++)
    {
        float k = generate(globalState, i) * 1;
        
        /*while(k > n*n-1)
        {
            k-=(n*n-1);
        }*/

        tabGauss[i] = k;
    }
}


int main() 
{
    int N=15;

    hiprandState* devStates;
    hipMalloc ( &devStates, N*sizeof( hiprandState ) );

    // setup seeds
    setup_kernel <<< 1, N >>> ( devStates,unsigned(time(NULL)) );

    float N2[N];
    float* N3;
    hipMalloc((void**) &N3, sizeof(float)*N);

    kernel<<<1,1>>> (N3, devStates, n, N);

    hipMemcpy(N2, N3, sizeof(float)*N, hipMemcpyDeviceToHost);

    for(int i=0;i<N;i++)
    {
        cout<<N2[i]<<endl;
    }



    return 0;
}