#include "cudaLib.h"


/*
*   Constructor for CudaLib
*/
CudaLib::CudaLib(){
	 int deviceCount;

    hipGetDeviceCount(&deviceCount);

	for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, dev);

        
    	
        if(dev == 0){
    		this->maxDevice = deviceProp.maxThreadsDim[dev];
        }
    }

}

/*
*   Destructor for CudaLib
*/
CudaLib::~CudaLib(){}

void CudaLib::loadOption(Option* option){

}

void CudaLib::loadBS(BS* bs){

}

void CudaLib::loadMonteCarlo(MonteCarlo* mc){

}
