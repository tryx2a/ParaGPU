#include "cudaLib.h"


/*
*   Constructor for CudaLib
*/
CudaLib::CudaLib(){
	 int deviceCount;

    hipGetDeviceCount(&deviceCount);

	for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, dev);

        
    	
        if(dev == 0){
    		this->maxDevice = deviceProp.maxThreadsDim[dev];
        }
    }

}

/*
*   Destructor for CudaLib
*/
CudaLib::~CudaLib(){}

void CudaLib::loadOption(Option* opt){
        hipError_t err;
      
        err = hipMalloc((void **)&(opt->T_),sizeof(float));
        err = hipMalloc((void **)&(opt->timeSteps_),sizeof(int));
        err = hipMalloc((void **)&(opt->size_),sizeof(int));
        
        err = hipMalloc((void **)&( (dynamic_cast<OptionBasket*>(opt))->strike_ ),sizeof(float));
        
        int sizePayoffCoeff = (dynamic_cast<OptionBasket*>(opt))->payoffCoeff_->size;
        double *arrayPayoffCoeff = (dynamic_cast<OptionBasket*>(opt))->payoffCoeff_->array;
        err = hipMalloc((void **)&(sizePayoffCoeff),sizeof(int));
        err = hipMalloc((void **)arrayPayoffCoeff ,sizeof(float)*sizePayoffCoeff);
     
}

void CudaLib::loadBS(BS* bs){

}

void CudaLib::loadMonteCarlo(MonteCarlo* mc){
        loadOption(mc->opt_);
        loadBS(mc->mod_);
        
        hipError_t err;
        err = hipMalloc((void **)&(this->h_),sizeof(float));
        err = hipMalloc((void **)&(this->H_),sizeof(int));
        err = hipMalloc((void **)&(this->samples_),sizeof(int));
}
