#include "cudaLib.h"


/*
*   Constructor for CudaLib
*/
CudaLib::CudaLib(MonteCarlo* mc){
  int deviceCount;

  hipGetDeviceCount(&deviceCount);

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;  
    hipGetDeviceProperties(&deviceProp, dev);
    if(dev == 0){
      this->maxDevice = deviceProp.maxThreadsDim[dev];
    }
  }
  
  this->allocMonteCarlo(mc);
}

/*
*   Destructor for CudaLib
*/
CudaLib::~CudaLib(){
  
}

void CudaLib::allocOption(Option* opt){
        hipError_t err;   
        
        //Allocation de la maturité
        err = hipMalloc((void **) &(this->T),sizeof(float));
        if(err != hipSuccess){
          printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
          exit(EXIT_FAILURE);
        }
        
        //Allocation du nombre de pas de constatation
        err = hipMalloc((void **) &(this->TimeSteps),sizeof(int));
        if(err != hipSuccess){
          printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
          exit(EXIT_FAILURE);
        }
        
        //Allocation du strike pour une option basket
        err = hipMalloc((void **) &(this->strike),sizeof(float));
        if(err != hipSuccess){
          printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
          exit(EXIT_FAILURE);
        }
        
        //Allocation de la taille du vecteur PayoffCoeff
        err = hipMalloc((void **)&(this->size_payoffCoeff),sizeof(int));
        if(err != hipSuccess){
          printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
          exit(EXIT_FAILURE);
        }
        
        //Allocation du tableau du vecteur PayoffCoeff
        err = hipMalloc((void **) &(this->payoffCoeff) ,sizeof(double)*opt->size_);
        if(err != hipSuccess){
          printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
          exit(EXIT_FAILURE);
        }
     
}

void CudaLib::allocBS(BS* bs){

    //Allocation de la taille de BS
    hipError_t err = hipMalloc( (void**) &(this->size), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //Allocation du taux
    err = hipMalloc( (void**) &(this->r), sizeof(float));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    // trend size
    err = hipMalloc( (void**) &(this->size_trend), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*trend array
    err = hipMalloc( (void**) &(this->trend), sizeof(float)*bs->size_);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*rho_
    err = hipMalloc( (void**) &(this->rho), sizeof(float));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*sigma_ size
    err = hipMalloc( (void**) &(this->size_sigma), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*sigma_ array
    err = hipMalloc( (void**) &(this->sigma), sizeof(float)*bs->size_);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*spot_ size
    err = hipMalloc( (void**) &(this->size_spot), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*spot_ array
    err = hipMalloc( (void**) &(this->spot), sizeof(float)*bs->size_);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*chol_ lig
    err = hipMalloc( (void**) &(this->m), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }
    
    //*chol_ col
    err = hipMalloc( (void**) &(this->n), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*chol_ array
    err = hipMalloc( (void**) &(this->chol), sizeof(float)*bs->chol->m*bs->chol->n);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }
}

void CudaLib::allocMonteCarlo(MonteCarlo* mc){
        
        hipError_t err;
        
        //Allocation du pas de différence fini     
        err = hipMalloc((void **) &(this->h),sizeof(float));
        if(err != hipSuccess){
          printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
          exit(EXIT_FAILURE);
        }
        
        //Allocation du nombre de date à couvrir
        err = hipMalloc((void **) &(this->H),sizeof(int));
        if(err != hipSuccess){
          printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
          exit(EXIT_FAILURE);
        }
        
        //Allocation du nombre de tour de la boucle MonteCarlo
        err = hipMalloc((void **) &(this->samples),sizeof(int));
        if(err != hipSuccess){
          printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
          exit(EXIT_FAILURE);
        }
        
        //Allocation de l'option
        allocOption(mc->opt_);
        //Allocation du modèle de Black&Scholes
        allocBS(mc->mod_);
             
}





