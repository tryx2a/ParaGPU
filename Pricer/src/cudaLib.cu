#include "cudaLib.h"


/*
*   Constructor for CudaLib
*/
CudaLib::CudaLib(){
	 int deviceCount;

    hipGetDeviceCount(&deviceCount);

	for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, dev);
    	
        if(dev == 0){
    		this->maxDevice = deviceProp.maxThreadsDim[dev];
        }
    }

}

/*
*   Destructor for CudaLib
*/
CudaLib::~CudaLib(){}

void CudaLib::loadOption(Option* option){

}

void CudaLib::loadBS(BS* bs){
    //Charge la taille de BS
    hipError_t err = hipMalloc( (void**) &(bs->size_), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //Charge le taux
    err = hipMalloc( (void**) &(bs->r_), sizeof(float));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    // trend size
    int sizeTrend = bs->trend->size;
    err = hipMalloc( (void**) &(sizeTrend), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*trend array
    double* arrayTrend = bs->trend->array;
    err = hipMalloc( (void**) arrayTrend, sizeof(float)*sizeTrend);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*rho_
    err = hipMalloc( (void**) &(bs->rho_), sizeof(float));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*sigma_ size
    int sizeSigma = bs->sigma_->size;
    err = hipMalloc( (void**) &(sizeSigma), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*sigma_ array
    double* arraySigma = bs->sigma_->array;
    err = hipMalloc( (void**) arraySigma, sizeof(float)*sizeSigma);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*spot_ size
    int sizeSpot = bs->spot_->size;
    err = hipMalloc( (void**) &(sizeSpot), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*spot_ array
    double* arraySpot = bs->spot_->array;
    err = hipMalloc( (void**) arraySpot, sizeof(float)*sizeSpot);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*chol_ lig
    int ligChol = bs->chol->m;
    err = hipMalloc( (void**) &(ligChol), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }
    
    //*chol_ col
    int colChol = bs->chol->n;
    err = hipMalloc( (void**) &(colChol), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*chol_ array
    double* arrayChol = bs->chol->array;
    err = hipMalloc( (void**) arrayChol, sizeof(double)*ligChol*colChol);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }
}

void CudaLib::loadMonteCarlo(MonteCarlo* mc){

}
