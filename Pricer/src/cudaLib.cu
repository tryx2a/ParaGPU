#include "cudaLib.h"


/*
*   Constructor for CudaLib
*/
CudaLib::CudaLib(){
	 int deviceCount;

    hipGetDeviceCount(&deviceCount);

	for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, dev);
    	
        if(dev == 0){
    		this->maxDevice = deviceProp.maxThreadsDim[dev];
        }
    }

}

/*
*   Destructor for CudaLib
*/
CudaLib::~CudaLib(){}

void CudaLib::loadOption(Option* opt){
        hipError_t err;
      
        err = hipMalloc((void **)&(opt->T_),sizeof(float));
        err = hipMalloc((void **)&(opt->timeSteps_),sizeof(int));
        err = hipMalloc((void **)&(opt->size_),sizeof(int));
        
        err = hipMalloc((void **)&( (dynamic_cast<OptionBasket*>(opt))->strike_ ),sizeof(float));
        
        int sizePayoffCoeff = (dynamic_cast<OptionBasket*>(opt))->payoffCoeff_->size;
        double *arrayPayoffCoeff = (dynamic_cast<OptionBasket*>(opt))->payoffCoeff_->array;
        err = hipMalloc((void **)&(sizePayoffCoeff),sizeof(int));
        err = hipMalloc((void **)arrayPayoffCoeff ,sizeof(float)*sizePayoffCoeff);
     
}

void CudaLib::loadBS(BS* bs){
    //Charge la taille de BS
    hipError_t err = hipMalloc( (void**) &(bs->size_), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //Charge le taux
    err = hipMalloc( (void**) &(bs->r_), sizeof(float));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    // trend size
    int sizeTrend = bs->trend->size;
    err = hipMalloc( (void**) &(sizeTrend), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*trend array
    double* arrayTrend = bs->trend->array;
    err = hipMalloc( (void**) arrayTrend, sizeof(float)*sizeTrend);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*rho_
    err = hipMalloc( (void**) &(bs->rho_), sizeof(float));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*sigma_ size
    int sizeSigma = bs->sigma_->size;
    err = hipMalloc( (void**) &(sizeSigma), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*sigma_ array
    double* arraySigma = bs->sigma_->array;
    err = hipMalloc( (void**) arraySigma, sizeof(float)*sizeSigma);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*spot_ size
    int sizeSpot = bs->spot_->size;
    err = hipMalloc( (void**) &(sizeSpot), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*spot_ array
    double* arraySpot = bs->spot_->array;
    err = hipMalloc( (void**) arraySpot, sizeof(float)*sizeSpot);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*chol_ lig
    int ligChol = bs->chol->m;
    err = hipMalloc( (void**) &(ligChol), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }
    
    //*chol_ col
    int colChol = bs->chol->n;
    err = hipMalloc( (void**) &(colChol), sizeof(int));
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }

    //*chol_ array
    double* arrayChol = bs->chol->array;
    err = hipMalloc( (void**) arrayChol, sizeof(double)*ligChol*colChol);
    if(err != hipSuccess){
      printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
      exit(EXIT_FAILURE);
    }
}

void CudaLib::loadMonteCarlo(MonteCarlo* mc){
        loadOption(mc->opt_);
        loadBS(mc->mod_);
        
        hipError_t err;
        err = hipMalloc((void **)&(this->h_),sizeof(float));
        err = hipMalloc((void **)&(this->H_),sizeof(int));
        err = hipMalloc((void **)&(this->samples_),sizeof(int));
}
