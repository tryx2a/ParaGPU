#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>

#include "../parser.h"
#include "../Method/mc.h"
#include "cudaLib.h"
#include "pricer_kernel.cuh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



int main(int argc, char ** argv) {

	const char *infile = argv[1];

	Param *P = new Parser(infile);
  MonteCarlo *mc = new MonteCarlo(P);

  //Creation du CudaLib
  CudaLib* cudaL = new CudaLib(mc);

  //Alloc des etats dans le GPU
  hiprandState* devStates;
  hipMalloc ( &devStates, (mc->samples_)*sizeof( hiprandState ) );
  
  /// Initialise la grille et les dimensions de chaque bloc
  dim3 DimGrid(mc->samples_/cudaL->maxDevice,1,1);
  dim3 DimBlock(cudaL->maxDevice,1,1);

  float strike = (dynamic_cast<OptionBasket *>(mc->opt_))->strike_;

  //Initialisation du noyau
  priceGPU <<<DimGrid, DimBlock>>>(cudaL->tabPrice, cudaL->tabIC, cudaL->tabPath, mc->mod_->size_, mc->mod_->r_, cudaL->spot, cudaL->sigma, cudaL->chol,
                                    mc->opt_->T_, mc->opt_->TimeSteps_, cudaL->payoffCoeff, strike, devStates, cudaL->maxDevice, unsigned(time(NULL)));

  
  float *priceTable = new float[mc->samples_];
  float *icTable = new float[mc->samples_];


  hipError_t err; 

  err = hipMemcpy(priceTable, cudaL->tabPrice, mc->samples_*sizeof(float), hipMemcpyDeviceToHost);
  if(err != hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(icTable, cudaL->tabIC, mc->samples_*sizeof(float), hipMemcpyDeviceToHost);
  if(err != hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__,__LINE__);
    exit(EXIT_FAILURE);
  }

 
  float prixReduction = 0.0;
  float varianceReduction = 0.0;
  float coeffActu = exp(-mc->mod_->r_*mc->opt_->T_);

  for(int i = 0; i<mc->samples_; i++){
      prixReduction += priceTable[i];
      varianceReduction += icTable[i];
  }

  prixReduction /= mc->samples_;
  varianceReduction /= mc->samples_;
  
  float varEstimator = exp(- 2 * (mc->mod_->r_ * mc->opt_->T_)) * (varianceReduction - (prixReduction*prixReduction));

  float prixFin = prixReduction*coeffActu;
  float ic = 2 * 1.96 * sqrt(varEstimator)/sqrt(mc->samples_);

  std::cout<<"Prix : "<<prixFin<<std::endl;
  std::cout<<"IC : "<<ic<<std::endl;


  free(priceTable);
  free(icTable);

  delete P;
  delete mc;

 
	return 0;
}
