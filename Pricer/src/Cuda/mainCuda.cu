#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>

#include "../parser.h"
#include "../Method/mc.h"
#include "cudaLib.h"
#include "pricer_kernel.cuh"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>



int main(int argc, char ** argv) {

	const char *infile = argv[1];

	Param *P = new Parser(infile);
  MonteCarlo *mc = new MonteCarlo(P);
  
  //Creation du CudaLib permettant d'allouer la mémoire nécessaire sur le GPU et faisant les copies
  CudaLib* cudaL = new CudaLib(mc);

  //Alloc des états dans le GPU
  hiprandState* devStates;
  hipMalloc ( &devStates, (mc->samples_)*sizeof( hiprandState ) );
  
  /// Initialise la grille et les dimensions de chaque bloc
  dim3 DimGrid(mc->samples_/cudaL->maxDevice,1,1);
  dim3 DimBlock(cudaL->maxDevice,1,1);
  
  //Appel du noyau
  priceGPU <<<DimGrid, DimBlock>>>(cudaL->tabPrice, cudaL->tabVar, cudaL->tabPath, mc->mod_->size_, mc->mod_->r_, cudaL->spot, cudaL->sigma, cudaL->chol,
                                    mc->opt_->T_, mc->opt_->TimeSteps_, cudaL->payoffCoeff, cudaL->lowerBarrier, cudaL->upperBarrier, cudaL->strike, 
                                    mc->opt_->id_, devStates, cudaL->maxDevice, unsigned(time(NULL)));


  /*
   * Réduction
   */ 

  //Constantes définissant la grille à utiliser pour l'allocation de la grille
  int num_elements = mc->samples_;
  size_t block_size = cudaL->maxDevice;
  size_t num_blocks = mc->samples_/cudaL->maxDevice;

  //Allocation des variables qui contiendront les résultats des réductions
  float *d_partial_sums_and_total_price;
  float *device_result_price;
  float *d_partial_sums_and_total_var;
  float *device_result_var;
  hipMalloc((void**)&d_partial_sums_and_total_price, sizeof(float) * num_blocks);
  hipMalloc((void**)&device_result_price, sizeof(float));
  hipMalloc((void**)&d_partial_sums_and_total_var, sizeof(float) * num_blocks);
  hipMalloc((void**)&device_result_var, sizeof(float));

  float payoffReduction = 0.0;
  float payoffSquareReduction = 0.0;

  int puissance = (int)(log(num_elements)/log(2));
  

  while( puissance >= 9){ // car 2^9 = 512

    // launch one kernel to compute, per-block, a partial sum
    block_sum<<<num_blocks,block_size,block_size * sizeof(float)>>>(cudaL->tabPrice + (mc->samples_ - num_elements), d_partial_sums_and_total_price, num_elements);
    block_sum<<<num_blocks,block_size,block_size * sizeof(float)>>>(cudaL->tabVar + (mc->samples_ - num_elements), d_partial_sums_and_total_var, num_elements);

    // launch a single block to compute the sum of the partial sums
    block_sum<<<1,num_blocks,num_blocks * sizeof(float)>>>(d_partial_sums_and_total_price, device_result_price, num_blocks);
    block_sum<<<1,num_blocks,num_blocks * sizeof(float)>>>(d_partial_sums_and_total_var, device_result_var, num_blocks);

    // copy the result back to the host
    float host_result_price = 0;
    float host_result_var = 0;
    hipMemcpy(&host_result_price, device_result_price, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&host_result_var, device_result_var, sizeof(float), hipMemcpyDeviceToHost);

    payoffReduction += host_result_price;
    payoffSquareReduction += host_result_var;

    num_elements -= (int)(pow(2.0,puissance));
    num_blocks = num_elements/cudaL->maxDevice;
    puissance = (int)(log(num_elements)/log(2));

  }

  // deallocate device memory
  hipFree(d_partial_sums_and_total_price);
  hipFree(device_result_price);
  hipFree(d_partial_sums_and_total_var);
  hipFree(device_result_var);

  payoffReduction /= mc->samples_;
  payoffSquareReduction /= mc->samples_;

  float coeffActu = exp(-mc->mod_->r_*mc->opt_->T_);
  float varEstimator = exp(- 2 * (mc->mod_->r_ * mc->opt_->T_)) * (payoffSquareReduction - (payoffReduction*payoffReduction));

  float prixFin = payoffReduction*coeffActu;
  float ic = 2 * 1.96 * sqrt(varEstimator)/sqrt(mc->samples_);

  std::cout<<"Prix : "<<prixFin<<std::endl;
  std::cout<<"IC : "<<ic<<std::endl;


  delete P;
  delete mc;
  delete cudaL;
 
	return 0;
}

